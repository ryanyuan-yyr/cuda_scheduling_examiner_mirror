#include "hip/hip_runtime.h"
// This file contains the implementation of the functions defined in
// task_host_utilities.h--used by task_host_utilities.c to work with the GPU.
#include <hip/hip_runtime.h>
#include <errno.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <sys/mman.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <time.h>
#include <unistd.h>
#include "task_host_utilities.h"
#include "library_interface.h"

// The number of GPU nanoseconds to spin for GetGPUTimerScale. Increasing this
// will both increase the accuracy and the time the function takes to return.
#define TIMER_SPIN_DURATION (2ull * 1000 * 1000 * 1000)

// This macro takes a hipError_t value. It prints an error message and returns
// 0 if the hipError_t isn't hipSuccess. Otherwise, it returns nonzero.
#define CheckCUDAError(val) (InternalCUDAErrorCheck((val), #val, __FILE__, __LINE__))

// Prints an error message and returns 0 if the given CUDA result is an error.
static int InternalCUDAErrorCheck(hipError_t result, const char *fn,
    const char *file, int line) {
  if (result == hipSuccess) return 1;
  printf("CUDA error %d: %s. In %s, line %d (%s)\n", (int) result,
    hipGetErrorString(result), file, line, fn);
  return 0;
}

static double CurrentSeconds(void) {
  struct timespec ts;
  if (clock_gettime(CLOCK_MONOTONIC_RAW, &ts) != 0) {
    printf("Error getting time.\n");
    exit(1);
  }
  return ((double) ts.tv_sec) + (((double) ts.tv_nsec) / 1e9);
}

// Returns the value of CUDA's global nanosecond timer.
static __device__ inline uint64_t GlobalTimer64(void) {
  // Due to a bug in CUDA's 64-bit globaltimer, the lower 32 bits can wrap
  // around after the upper bits have already been read. Work around this by
  // reading the high bits a second time. Use the second value to detect a
  // rollover, and set the lower bits of the 64-bit "timer reading" to 0, which
  // would be valid, it's passed over during the duration of the reading. If no
  // rollover occurred, just return the initial reading.
  volatile uint64_t first_reading;
  volatile uint32_t second_reading;
  uint32_t high_bits_first;
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(first_reading));
  high_bits_first = first_reading >> 32;
  asm volatile("mov.u32 %0, %%globaltimer_hi;" : "=r"(second_reading));
  if (high_bits_first == second_reading) {
    return first_reading;
  }
  // Return the value with the updated high bits, but the low bits set to 0.
  return ((uint64_t) second_reading) << 32;
}

// A simple kernel which writes the value of the globaltimer64 register to a
// location in device memory.
static __global__ void GetTime(uint64_t *time) {
  *time = GlobalTimer64();
}

// Allocates a private shared memory buffer containing the given number of
// bytes. Can be freed by using FreeSharedBuffer. Returns NULL on error.
// Initializes the buffer to contain 0.
static void* AllocateSharedBuffer(size_t size) {
  void *to_return = mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_ANONYMOUS |
    MAP_SHARED, -1, 0);
  if (to_return == MAP_FAILED) return NULL;
  memset(to_return, 0, size);
  return to_return;
}

// Frees a shared buffer returned by AllocateSharedBuffer.
static void FreeSharedBuffer(void *buffer, size_t size) {
  munmap(buffer, size);
}

// This function should be run in a separate process in order to read the GPU's
// nanosecond counter. Returns 0 on error.
static void InternalReadGPUNanoseconds(int cuda_device, double *cpu_time,
    uint64_t *gpu_time) {
  uint64_t *device_time = NULL;
  if (!CheckCUDAError(hipSetDevice(cuda_device))) return;
  if (!CheckCUDAError(hipMalloc(&device_time, sizeof(*device_time)))) return;
  // Run the kernel a first time to warm up the GPU.
  GetTime<<<1, 1>>>(device_time);
  if (!CheckCUDAError(hipDeviceSynchronize())) return;
  // Now run the actual time-checking kernel.
  GetTime<<<1, 1>>>(device_time);
  *cpu_time = CurrentSeconds();
  if (!CheckCUDAError(hipMemcpy(gpu_time, device_time, sizeof(*gpu_time),
    hipMemcpyDeviceToHost))) {
    hipFree(device_time);
    return;
  }
  hipFree(device_time);
}

extern "C" int GetHostDeviceTimeOffset(int cuda_device, double *host_seconds,
  uint64_t *gpu_nanoseconds) {
  uint64_t *shared_gpu_time = NULL;
  double *shared_cpu_time = NULL;
  int status;
  pid_t pid = -1;
  shared_gpu_time = (uint64_t *) AllocateSharedBuffer(
    sizeof(*shared_gpu_time));
  if (!shared_gpu_time) {
    printf("Failed allocating shared buffer for IPC.\n");
    return 0;
  }
  shared_cpu_time = (double *) AllocateSharedBuffer(sizeof(*shared_cpu_time));
  if (!shared_cpu_time) {
    printf("Failed allocating shared CPU time buffer for IPC.\n");
    FreeSharedBuffer(shared_gpu_time, sizeof(*shared_gpu_time));
    return 0;
  }
  pid = fork();
  if (pid < 0) {
    printf("Failed creating a child process to get GPU time: %s\n", strerror(
      errno));
    return 0;
  }
  if (pid == 0) {
    // The following CUDA code is run in the child process
    InternalReadGPUNanoseconds(cuda_device, shared_cpu_time, shared_gpu_time);
    exit(0);
  }
  // The parent will wait for the child to finish, then return the value
  // written to the shared buffer.
  if (wait(&status) < 0) {
    printf("Failed waiting on the child process.\n");
    FreeSharedBuffer(shared_cpu_time, sizeof(*shared_cpu_time));
    FreeSharedBuffer(shared_gpu_time, sizeof(*shared_gpu_time));
    return 0;
  }
  *host_seconds = *shared_cpu_time;
  *gpu_nanoseconds = *shared_gpu_time;
  FreeSharedBuffer(shared_cpu_time, sizeof(*shared_cpu_time));
  FreeSharedBuffer(shared_gpu_time, sizeof(*shared_gpu_time));
  if (!WIFEXITED(status)) {
    printf("The child process didn't exit normally.\n");
    return 0;
  }
  return 1;
}

// This function should always be run in a separate process.
static int InternalGetMaxResidentThreads(int cuda_device) {
  struct hipDeviceProp_t properties;
  int warps_per_sm = 64;
  if (!CheckCUDAError(hipGetDeviceProperties(&properties, cuda_device))) {
    return 0;
  }
  // Compute capability 2.0 devices have a 48 warps per SM.
  if (properties.major <= 2) warps_per_sm = 48;
  return warps_per_sm * properties.multiProcessorCount * properties.warpSize;
}

extern "C" int GetMaxResidentThreads(int cuda_device) {
  int to_return, status;
  pid_t pid = -1;
  int *max_thread_count = NULL;
  max_thread_count = (int *) AllocateSharedBuffer(sizeof(*max_thread_count));
  if (!max_thread_count) {
    printf("Failed allocating shared buffer for IPC.\n");
    return 0;
  }
  pid = fork();
  if (pid < 0) {
    printf("Failed creating a child process to get thread count: %s\n",
      strerror(errno));
    return 0;
  }
  if (pid == 0) {
    // The following CUDA code is run in the child process
    *max_thread_count = InternalGetMaxResidentThreads(cuda_device);
    exit(0);
  }
  // The parent will wait for the child to finish, then return the value
  // written to the shared buffer.
  if (wait(&status) < 0) {
    printf("Failed waiting on the child process.\n");
    FreeSharedBuffer(max_thread_count, sizeof(*max_thread_count));
    return 0;
  }
  to_return = *max_thread_count;
  FreeSharedBuffer(max_thread_count, sizeof(*max_thread_count));
  if (!WIFEXITED(status)) {
    printf("The child process didn't exit normally.\n");
    return 0;
  }
  return to_return;
}

static __global__ void TimerSpin(uint64_t ns_to_spin) {
  uint64_t start_time = GlobalTimer64();
  while ((GlobalTimer64() - start_time) < ns_to_spin) {
    continue;
  }
}

// This function is intended to be run in a child process. Returns -1 on error.
static double InternalGetGPUTimerScale(int cuda_device) {
  struct timespec start, end;
  uint64_t nanoseconds_elapsed;
  if (!CheckCUDAError(hipSetDevice(cuda_device))) return -1;
  // Run the kernel once to warm up the GPU.
  TimerSpin<<<1, 1>>>(1000);
  if (!CheckCUDAError(hipDeviceSynchronize())) return -1;
  // After warming up, do the actual timing.
  if (clock_gettime(CLOCK_MONOTONIC_RAW, &start) != 0) {
    printf("Failed getting start time.\n");
    return -1;
  }
  TimerSpin<<<1, 1>>>(TIMER_SPIN_DURATION);
  if (!CheckCUDAError(hipDeviceSynchronize())) return -1;
  if (clock_gettime(CLOCK_MONOTONIC_RAW, &end) != 0) {
    printf("Failed getting end time.\n");
    return -1;
  }
  nanoseconds_elapsed = end.tv_sec * 1e9 + end.tv_nsec;
  nanoseconds_elapsed -= start.tv_sec * 1e9 + start.tv_nsec;
  return ((double) nanoseconds_elapsed) / ((double) TIMER_SPIN_DURATION);
}

extern "C" double GetGPUTimerScale(int cuda_device) {
  double to_return;
  double *scale = NULL;
  int status;
  pid_t pid;
  scale = (double *) AllocateSharedBuffer(sizeof(*scale));
  if (!scale) {
    printf("Failed allocating space to hold the GPU time scale.\n");
    return -1;
  }
  pid = fork();
  if (pid < 0) {
    printf("Failed creating a child process.\n");
    FreeSharedBuffer(scale, sizeof(*scale));
    return -1;
  }
  if (pid == 0) {
    // Access the GPU with the child process only.
    *scale = InternalGetGPUTimerScale(cuda_device);
    exit(0);
  }
  if (wait(&status) < 0) {
    printf("Failed waiting on the child process.\n");
    FreeSharedBuffer(scale, sizeof(*scale));
    return -1;
  }
  to_return = *scale;
  FreeSharedBuffer(scale, sizeof(*scale));
  if (!WIFEXITED(status)) {
    printf("The child process didn't exit normally.\n");
    return -1;
  }
  return to_return;
}

extern "C" int GetSingleBlockAndGridDimensions(InitializationParameters *params,
    int *thread_count, int *block_count) {
  int a, b;
  if ((params->block_dim[1] != 1) || (params->block_dim[2] != 1)) {
    printf("Expected 1-D block dimensions, but got [%d, %d, %d]\n",
      params->block_dim[0], params->block_dim[1], params->block_dim[2]);
    return 0;
  }
  if ((params->grid_dim[1] != 1) || (params->grid_dim[2] != 1)) {
    printf("Expected 1-D grid dimensions, but got [%d, %d, %d]\n",
      params->grid_dim[0], params->grid_dim[1], params->grid_dim[2]);
    return 0;
  }
  a = params->block_dim[0];
  if ((a < 1) || (a > 1024)) {
    printf("Invalid number of threads in a block: %d\n", a);
    return 0;
  }
  b = params->grid_dim[0];
  if (b < 1) {
    printf("Invalid number of blocks: %d\n", b);
  }
  *thread_count = a;
  *block_count = b;
  return 1;
}

extern "C" int GetSingleBlockDimension(InitializationParameters *params,
    int *thread_count) {
  int x, y, z;
  x = params->block_dim[0];
  y = params->block_dim[1];
  z = params->block_dim[2];
  if ((y != 1) || (z != 1)) {
    printf("Expected 1-D block dimensions, but got [%d, %d, %d]\n", x, y, z);
    return 0;
  }
  *thread_count = x;
  return 1;
}
